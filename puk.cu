
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>

__global__
void puk(char *dst, char *src, int wid, int hei)
{
    int stride = gridDim.x * blockDim.x,
        idx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int di = idx; di < wid * hei; di += stride)
    {
        int dxi, dyi, sxi, syi;
        float dxf, dyf, sxf, syf;

        dxi = di % wid;
        dyi = di / wid;
        dxf = (float )(dxi - wid/2) / (wid/2);
        dyf = (float )(dyi - hei/2) / (hei/2);
        sxf = dxf<0 ? -dxf*dxf : dxf*dxf;
        syf = dyf<0 ? -dyf*dyf : dyf*dyf;
        sxi = sxf * wid/2 + wid/2;
        syi = syf * hei/2 + hei/2;

        int si = wid * syi + sxi;
        dst[di*3+0] = src[si*3+0];
        dst[di*3+1] = src[si*3+1];
        dst[di*3+2] = src[si*3+2];
    }
}

__global__
void unpuk(char *dst, char *src, int wid, int hei)
{
    int stride = gridDim.x * blockDim.x,
        idx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int di = idx; di < wid * hei; di += stride)
    {
        int dxi, dyi, sxi, syi;
        float dxf, dyf, sxf, syf;

        dxi = di % wid;
        dyi = di / wid;
        dxf = (float )(dxi - wid/2) / (wid/2);
        dyf = (float )(dyi - hei/2) / (hei/2);
        sxf = dxf<0 ? -sqrt(-dxf) : sqrt(dxf);
        syf = dyf<0 ? -sqrt(-dyf) : sqrt(dyf);
        sxi = sxf * wid/2 + wid/2;
        syi = syf * hei/2 + hei/2;

        int si = wid * syi + sxi;
        dst[di*3+0] = src[si*3+0];
        dst[di*3+1] = src[si*3+1];
        dst[di*3+2] = src[si*3+2];
    }
}

int main(int argc, char **argv)
{
    char *buf, *_sbuf, *_dbuf;
    size_t sz;
    uint32_t off, wid, hei;

    fseek(stdin, 0, SEEK_END);
    sz = ftell(stdin);
    rewind(stdin);
    buf = (char *)malloc(sz);
    fread(buf, 1, sz, stdin);

    off = *(uint32_t *)&buf[0xa];
    wid = *(uint32_t *)&buf[0x12];
    hei = *(uint32_t *)&buf[0x16];

    hipMalloc(&_sbuf, sz);
    hipMemcpy(_sbuf, buf, sz, hipMemcpyHostToDevice);

    hipMalloc(&_dbuf, sz);
    hipMemcpy(_dbuf, buf, sz, hipMemcpyHostToDevice);

    if (argc == 1)
        puk<<<0x100, 0x100>>>(_dbuf + off, _sbuf + off, wid, hei);
    else
        unpuk<<<0x100, 0x100>>>(_dbuf + off, _sbuf + off, wid, hei);
    hipDeviceSynchronize();

    hipMemcpy(buf, _dbuf, sz, hipMemcpyDeviceToHost);

    fwrite(buf, 1, sz, stdout);
}
